#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "hiprand.h"
#include  <iostream>
#include <chrono>
#include <stdio.h>
#include "Dummy.cuh"


__global__ void incrementDummy(Dummy *a)
{
	auto i = threadIdx.x + blockIdx.x * blockDim.x;
	Dummy myDummy;
	myDummy.incrementCounterDevice();
	a[i].counter = myDummy.counter;
}
int main()
{
	auto constexpr arraySize = 10000000;
	Dummy *arrPtr = new Dummy[arraySize];
	auto start = std::chrono::high_resolution_clock::now();
	incrementDummy <<< (arraySize + 255) / 256, 256 >>> (arrPtr);
	hipDeviceSynchronize();
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed1 = finish - start;
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		std::cout << hipGetErrorString(err) << std::endl;
	auto start2 = std::chrono::high_resolution_clock::now();
	for (auto i = 0; i < arraySize; ++i)
	{
		arrPtr[i].incrementCounterHostHost();
	}
	auto finish2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed2 = finish2 - start2;
	for(auto i = 0; i < arraySize; ++i)
	{
		if(arrPtr[i].counter != 2)
		{
			std::cout << arrPtr[i].counter << std::endl;
		}
	}
	std::cout << "Elapsed time: " << elapsed1.count() << " s\n";
	std::cout << "Elapsed time: " << elapsed2.count() << " s\n";
	delete arrPtr;
	return 0;
}