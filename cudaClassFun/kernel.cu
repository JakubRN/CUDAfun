#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include  <iostream>
#include <chrono>
#include <stdio.h>
class Managed {
public:
	void *operator new(size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}
	void *operator new[](size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}

	void operator delete(void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
	void operator delete[](void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
};

class Dummy : public Managed
{
public:
	int counter;
	Dummy() : counter(0) { ; }
	__device__ void incrementCounter() { ++counter; }
};

__global__ void incrementDummy(Dummy *a)
{
	auto i = threadIdx.x + blockIdx.x * blockDim.x;
	a[i].incrementCounter();
}
int main()
{
    auto constexpr arraySize = 500000000;
	Dummy *arrPtr = new Dummy[arraySize];
	auto start = std::chrono::high_resolution_clock::now();
	incrementDummy <<< (arraySize + 255) / 256, 256 >>> (arrPtr);
	hipDeviceSynchronize();
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed1 = finish - start;
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		std::cout << hipGetErrorString(err) << std::endl;
	auto start2 = std::chrono::high_resolution_clock::now();
	for (auto i = 0; i < arraySize; ++i)
	{
		arrPtr[i].counter++;
	}
	auto finish2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed2 = finish2 - start2;
	for(auto i = 0; i < arraySize; ++i)
	{
		if(arrPtr[i].counter != 2)
		{
			std::cout << arrPtr[i].counter << std::endl;
		}
	}
	std::cout << "Elapsed time: " << elapsed1.count() << " s\n";
	std::cout << "Elapsed time: " << elapsed2.count() << " s\n";
	delete arrPtr;
    return 0;
}