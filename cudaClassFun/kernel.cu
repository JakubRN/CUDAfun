#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "hiprand.h"
#include  <iostream>
#include <chrono>
#include <stdio.h>
class Managed {
public:
	void *operator new(size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}
	void *operator new[](size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}

	void operator delete(void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
	void operator delete[](void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
};

class Dummy : public Managed
{
public:
	int counter;
	double magicNumber;
	Dummy() : counter(0) { ; }
	__device__ void incrementCounter()
	{
		++counter;
		magicNumber = counter * 5 * pow((double)3, __double2int_rd(magicNumber) % 100) + counter * magicNumber + counter * 2 * pow((double)2, __double2int_rd(magicNumber) % 50) + counter * magicNumber;
	}
	void incrementHost()
	{
		++counter;
		magicNumber = counter * 5 * pow(3, ((int)magicNumber % 100)) + counter * magicNumber + counter * 2 * pow(2, ((int)magicNumber % 50)) + counter * magicNumber;
	}
};

__global__ void incrementDummy(Dummy *a)
{
	auto i = threadIdx.x + blockIdx.x * blockDim.x;
	a[i].incrementCounter();
}
int main()
{
    auto constexpr arraySize = 10000000;
	Dummy *arrPtr = new Dummy[arraySize];
	auto start = std::chrono::high_resolution_clock::now();
	incrementDummy <<< (arraySize + 255) / 256, 256 >>> (arrPtr);
	hipDeviceSynchronize();
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed1 = finish - start;
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		std::cout << hipGetErrorString(err) << std::endl;
	auto start2 = std::chrono::high_resolution_clock::now();
	for (auto i = 0; i < arraySize; ++i)
	{
		arrPtr[i].incrementHost();
	}
	auto finish2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> elapsed2 = finish2 - start2;
	for(auto i = 0; i < arraySize; ++i)
	{
		if(arrPtr[i].counter != 2)
		{
			std::cout << arrPtr[i].counter << std::endl;
		}
	}
	std::cout << "Elapsed time: " << elapsed1.count() << " s\n";
	std::cout << "Elapsed time: " << elapsed2.count() << " s\n";
	delete arrPtr;
    return 0;
}