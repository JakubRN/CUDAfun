#include "hip/hip_runtime.h"
#include "Dummy.cuh"
#include <cmath>
__host__ __device__ Dummy::Dummy() : counter(0) { ; }
__device__ void Dummy::incrementCounterDevice()
{
	++counter;
	magicNumber = counter * 5 * pow((double)3, __double2int_rd(magicNumber) % 100) + counter * magicNumber + counter * 2 * pow((double)2, __double2int_rd(magicNumber) % 50) + counter * magicNumber;
}
__host__ void Dummy::incrementCounterHostHost()
{
	++counter;
	magicNumber = counter * 5 * pow(3, ((int)magicNumber % 100)) + counter * magicNumber + counter * 2 * pow(2, ((int)magicNumber % 50)) + counter * magicNumber;
}